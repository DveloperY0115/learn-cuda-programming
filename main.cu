
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d, %d] From device\n", threadIdx.x, blockIdx.x);
}

int main() {
    std::cout << "Hello, World! from host!" << std::endl;
    print_from_gpu<<<1,2>>>();
    hipDeviceSynchronize();
    return 0;
}
