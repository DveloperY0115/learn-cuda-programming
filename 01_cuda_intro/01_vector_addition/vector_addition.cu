//
// Created by dveloperY0115 on 1/6/2021.
//


#include <hip/hip_runtime.h>
#include <iostream>

#define N 512

/**
 * Takes three arrays of int, performs vector addition and saves the result
 * @param a operand 1
 * @param b operand 2
 * @param c the array where results will be stored in
 */
void host_add(const int* a, const int* b, int* c) {
    for (int idx = 0; idx < N; idx++)
        c[idx] = a[idx] + b[idx];
}

/**
 * Fills the elements of array with its positional indices
 * @param data array of int
 */
void fill_array(int *data) {
    for (int idx = 0; idx < N; idx++)
        data[idx] = idx;
}

/**
 * Takes three arrays of int, prints the arithmetic relationship (addition) of these
 * @param a array 1
 * @param b array 2
 * @param c array 3 (result)
 */
void print_output(const int* a, const int* b, const int* c) {
    for (int idx = 0; idx < N; idx++)
        std::cout << a[idx] << " + " << b[idx] << " = " << c[idx] << std::endl;
}

int main() {
    int *a, *b, *c;
    int size = N * sizeof(int);

    a = (int*) malloc(size); fill_array(a);
    b = (int*) malloc(size); fill_array(b);
    c = (int*) malloc(size);
    host_add(a, b, c);
    print_output(a, b, c);
    free(a); free(b); free(c);
    return 0;
}