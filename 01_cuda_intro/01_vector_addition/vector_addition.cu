//
// Created by dveloperY0115 on 1/6/2021.
//


#include <hip/hip_runtime.h>
#include <iostream>

#define N 512

/**
 * Takes three arrays of int, performs vector addition and saves the result
 * @param a operand 1
 * @param b operand 2
 * @param c the array where results will be stored in
 */
void host_add(const int* a, const int* b, int* c) {
    for (int idx = 0; idx < N; idx++)
        c[idx] = a[idx] + b[idx];
}

__global__ void device_add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}

/**
 * Fills the elements of array with its positional indices
 * @param data array of int
 */
void fill_array(int *data) {
    for (int idx = 0; idx < N; idx++)
        data[idx] = idx;
}

/**
 * Takes three arrays of int, prints the arithmetic relationship (addition) of these
 * @param a array 1
 * @param b array 2
 * @param c array 3 (result)
 */
void print_output(const int* a, const int* b, const int* c) {
    for (int idx = 0; idx < N; idx++)
        std::cout << a[idx] << " + " << b[idx] << " = " << c[idx] << std::endl;
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;   // device copies of a, b, c
    int size = N * sizeof(int);

    // allocate memory on host
    a = (int*) malloc(size); fill_array(a);
    b = (int*) malloc(size); fill_array(b);
    c = (int*) malloc(size);

    // allocate memory on device
    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_c, N * sizeof(int));

    // copy host -> device
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // perform addition on device
    size_t threads_per_block = 8;
    size_t no_of_blocks = N / threads_per_block;
    device_add<<< no_of_blocks, threads_per_block >>>(d_a, d_b, d_c);

    // copy host <- device
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    print_output(a, b, c);

    // free host memory
    free(a); free(b); free(c);

    // free device memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}